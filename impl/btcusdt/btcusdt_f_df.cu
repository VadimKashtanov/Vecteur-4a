#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../../impl_template/tmpl_etc.cu"

static __global__ void k__f_df_btcusdt(
	float * S,
	//
	float * y, float * dy,
	float * w,
	//
	uint * ts__d,
	//
	uint I, uint T, uint L, uint N)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x; 
	//uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//uint  i = threadIdx.z + blockIdx.z * blockDim.z;
	//
	if (_y < (L*N)/* && _t < T *//*&& i < I*/) {
		float s = 0;
		FOR(0, i, I) {
			FOR(0, _t, GRAND_T) {
				FOR(0, mega_t, MEGA_T) {
					uint ty        = t_MODE(_t, mega_t);
					uint t_btcusdt = ts__d[_t] + 1 + mega_t;
					assert(t_btcusdt < T);
					//
					float __y = y[ty*I*L*N + i*L*N + _y];
					float __w = w[i*T*L*N  + t_btcusdt*L*N  +  _y];
					assert(__y >= -100 && __y <= +100);
					//
					float coef = (float)(GRAND_T * MEGA_T * (I*L*N));
					s       += ( score_p2(__y, __w, 2));
					float ds = (dscore_p2(__y, __w, 2)) / coef;
					//
					atomicAdd(&dy[ty*I*N*L + i*L*N + _y], ds);
				}
			}
		}
		//
		atomicAdd(&S[0], s);
	}
};

/*
A fair :
	1) Finir d'ajouter les jour, mois année
	2) Tester le .T dar.bin
	3) ajouter le module de séparation des I (et embede et positionnal)
	4) Ajouter une union 4
*/

float f_df_btcusdt(BTCUSDT_t * btcusdt, float * y__d, float * dy__d, uint * ts__d) {
	uint I=btcusdt->I;
	uint L=btcusdt->L;
	uint N=btcusdt->N;
	uint T=btcusdt->T;
	//
	float * S__d = cudalloc<float>(1);
	k__f_df_btcusdt<<<dim3(KERD((L*N), 8)/*,KERD(GRAND_T, 8), *//*KERD(I,4)*/), dim3(8/*,8,*//*4*/)>>>(
		S__d,
		y__d, dy__d,
		btcusdt->serie__d,
		ts__d,
		btcusdt->I, btcusdt->T, btcusdt->L, btcusdt->N
	);
	ATTENDRE_CUDA();
	//
	//
	float * S = gpu_vers_cpu<float>(S__d, 1);
	//
	float coef = (float)(GRAND_T * MEGA_T * (I*L*N));
	float score = S[0]/coef;// / ((float)(MEGA_T * btcusdt->I * btcusdt->L * btcusdt->N));
	//
	//
	cudafree<float>(S__d);
	    free       (S   );
	//
	return score;
};
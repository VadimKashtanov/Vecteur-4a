#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../../impl_template/tmpl_etc.cu"

static __global__ void k__pourcent_btcusdt_stricte(
	float * inconnue_____somme,
	float *   connue_____somme,
	//
	float * y, float * w,
	uint * ts__d,
	//
	uint i,	//	Interv
	//
	uint I, uint T, uint L, uint N)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y%N!=0 && _y < L*N && _t < GRAND_T) {
		FOR(0, mega_t, MEGA_T) {
			uint ty        = t_MODE(_t, mega_t);
			uint t_btcusdt = ts__d[_t] + 1 + mega_t;
			//
			uint wpos = i*T*L*N + t_btcusdt*L*N + _y;
			//
			float delat_w = w[wpos     ] / w[wpos     -1] - 1;
			float delta_y = y[ty*I*L*N + i*L*N + _y] / y[ty*I*L*N + i*L*N + _y-1] - 1;
			//
			float a_t_il_predit = (float)(sng(delat_w) == sng(delta_y));
			//
			if (_y % N == N-1) atomicAdd(&inconnue_____somme[i], a_t_il_predit);
			else               atomicAdd(&  connue_____somme[i], a_t_il_predit);
		}
	}
};

float* pourcent_btcusdt(BTCUSDT_t * btcusdt, float * y__d, uint * ts__d) {
	uint I=btcusdt->I;
	uint L=btcusdt->L;
	uint N=btcusdt->N;
	uint T=btcusdt->T;
	//
	float * inconnue_____somme__d = cudalloc<float>(btcusdt->I);
	float *   connue_____somme__d = cudalloc<float>(btcusdt->I);
	//
	FOR(0, i, btcusdt->I) {
		k__pourcent_btcusdt_stricte<<<dim3(KERD((L*N), 16), KERD(GRAND_T, 16)), dim3(16,16)>>>(
			inconnue_____somme__d,
			  connue_____somme__d,
			y__d, btcusdt->serie__d,
			ts__d,
			i,
			btcusdt->I, btcusdt->T, btcusdt->L, btcusdt->N
		);
	}
	ATTENDRE_CUDA();
	//
	float * inconnue_____somme = gpu_vers_cpu<float>(inconnue_____somme__d, btcusdt->I);
	float *   connue_____somme = gpu_vers_cpu<float>(  connue_____somme__d, btcusdt->I);
	//
	float * ret = alloc<float>(btcusdt->I * 2);
	FOR(0, i, btcusdt->I) {
		inconnue_____somme[i] = inconnue_____somme[i] / (float)(MEGA_T*I*(L*N-1-1));
		  connue_____somme[i] =   connue_____somme[i] / (float)(MEGA_T*I*(+1     ));
		//
		ret[0*btcusdt->I + i] = inconnue_____somme[i];
		ret[1*btcusdt->I + i] =   connue_____somme[i];
	}
	//
	cudafree<float>(inconnue_____somme__d);
	cudafree<float>(  connue_____somme__d);
	    free       (inconnue_____somme   );
	    free       (  connue_____somme   );
	//
	return ret;
};

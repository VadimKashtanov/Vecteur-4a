#include "hip/hip_runtime.h"
#include "sortie.cuh"

__global__
static void kerd__sortie(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t-x0_t, mega_t);
		uint ty  = t_MODE(_t,      mega_t);
		//
		y[ty*Y + _y] = x0[tx0*X0 + _y];
	};
};

void sortie__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t) {
	uint Y = inst->Y;
	//
	kerd__sortie<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
		inst->x_t[0], inst->x_Y[0], x__d[0],
		//
		inst->Y,
		inst->y__d,
		//
		ts__d, mega_t
	);
};